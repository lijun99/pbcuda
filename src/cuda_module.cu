#include "hip/hip_runtime.h"
#include "cuda_module.h"
#include <iostream>

template <typename T>
__global__ void addKernel(T* c, const T* a, const T* b, const int size) {
    auto i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        c[i] = a[i] + b[i];
    }
}

template <typename T>
void cuda_add(T* c, const T* a, const T* b, int size) {
    T* d_a, * d_b, * d_c;
    size_t bytes = size * sizeof(T);
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);
    
    hipMemcpy(d_a, a, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, bytes, hipMemcpyHostToDevice);
    
    int threadsPerBlock = 256;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
    addKernel<<<blocksPerGrid, threadsPerBlock>>>(d_c, d_a, d_b, size);
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::cerr << "Error after hipSetDevice: " << hipGetErrorString(error) << std::endl;
    }
    
    hipMemcpy(c, d_c, bytes, hipMemcpyDeviceToHost);
    
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}

// **Explicit Instantiations: required for python module**
template void cuda_add<float>(float* c, const float* a, const float* b, int size);
template void cuda_add<double>(double* c, const double* a, const double* b, int size);
template void cuda_add<int>(int* c, const int* a, const int* b, int size);
template void cuda_add<int64_t>(int64_t* c, const int64_t* a, const int64_t* b, int size);